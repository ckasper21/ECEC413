#include "hip/hip_runtime.h"
/* Blur filter. Device code. */

#ifndef _BLUR_FILTER_KERNEL_H_
#define _BLUR_FILTER_KERNEL_H_

#include "blur_filter.h"

__global__ void 
blur_filter_kernel (const float *in, float *out, int size)
{
	int pix, i, j;
	int row, col;
	int curr_row, curr_col;
	float blur_value;
	int num_neighbors;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (pix = idx; pix < size * size; pix+=stride) {
		row = pix/size;
		col = pix % size;

		blur_value = 0.0;
		num_neighbors = 0;

		for (i = -BLUR_SIZE; i < (BLUR_SIZE + 1); i++) {
			for (j = -BLUR_SIZE; j < (BLUR_SIZE + 1); j++) {
				curr_row = row + i;
				curr_col = col + j;

				if ((curr_row > -1) && (curr_row < size) &&\
						(curr_col > -1) && (curr_col < size)) {
							blur_value += in[curr_row * size + curr_col];
							num_neighbors++;
						}
			}
		}

		out[pix] = blur_value / num_neighbors;
	}
	
	return;
}

#endif /* _BLUR_FILTER_KERNEL_H_ */